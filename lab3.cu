#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#define bsunpadded 8
#define bs 8

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }


double find_checksum(double * output)
{
    double checksum = 0.0;
    int K=10, H=4096, W=4096;
    for (int k=0; k<K; k++){
        for(int row=0; row<H; row++){
            for(int col=0; col<W; col++){
                checksum+=output[(k*H*W)+(row*W)+col];
            }
        }
    }
    return checksum;
}

__global__ void C1(int W, int H, int C, int FW, int FH, int K, double* input, double* filter, double* output) {

    int image_col = blockDim.x * blockIdx.x + threadIdx.x;
    int image_row = blockDim.y * blockIdx.y + threadIdx.y;
    int fil_num = blockIdx.z;

    if (image_row < H && image_col < W && fil_num < K) {

        int p1col = image_col - FW / 2;
        int p1row = image_row - FH / 2;
        double conv_val = 0.0, img_val = 0.0, fil_val = 0.0;
        int ncol = 0, nrow = 0;

        for (int c = 0; c < C; c++) {
            for (int row = 0; row < FH; row++) {
                for (int col = 0; col < FW; col++) {
                    img_val = 0.0;
                    ncol = p1col + col;
                    nrow = p1row + row;

                    fil_val = filter[(fil_num * C * FW * FH) + (c * FW * FH) + ((FH - row - 1) * FW) + (FW - 1 - col)];
                    if (0 <= ncol && ncol < W)
                        if (0 <= nrow && nrow < H)
                            img_val = input[(c * H * W) + (nrow) * W + ncol];

                    conv_val = conv_val + (img_val * fil_val);
                }
            }
        }
        output[(fil_num * H * W) + (image_row * W) + image_col] = conv_val;
    }
}

__global__ void C2(int W, int H, int C, int FW, int FH, int K, double* input, double* filter, double* output){

    int image_col = threadIdx.x + blockDim.x * blockIdx.x;
    int image_row = threadIdx.y + blockDim.y * blockIdx.y;
    int fil_num = threadIdx.z;

    __shared__ double tile[3][10][10];

    if(image_row<H && image_col<W && fil_num<K){

        int actual_tr = threadIdx.y, actual_tc = threadIdx.x;

        for(int c=0; c<C; c++){
            tile[c][actual_tr+1][actual_tc+1]=input[(c*W*H)+(image_row*W)+image_col];
        }

        if(actual_tr == bsunpadded - 1){
            for (int c = 0; c < C; c++){
                if (image_row < H - 1)
                    tile[c][actual_tr + 2][actual_tc + 1] = input[(c * H * W) + ((image_row + 1) * W) + image_col];
                else
                    tile[c][actual_tr + 2][actual_tc + 1] = 0.0;
            }


            if (actual_tc == bsunpadded - 1){
                for (int c = 0; c < C; c++) {
                    if (image_col < W - 1 && image_row < H - 1)
                        tile[c][actual_tr + 2][actual_tc + 2] = input[(c * H * W) + ((image_row + 1) * W) +
                                                                      image_col + 1];
                    else
                        tile[c][actual_tr + 2][actual_tc + 2] = 0.0;
                    }
                }
            }

        if(actual_tc == 0){
            for(int c=0;c<C;c++){
               if(image_col>=1)
                    tile[c][actual_tr+1][actual_tc] = input[(c*H*W)+(image_row*W)+image_col-1] ;
                else
                    tile[c][actual_tr+1][actual_tc] = 0.0 ;
            }

            if(actual_tr == bsunpadded - 1){
                    for(int c=0;c<C;c++) {
                        if (image_col >= 1 && image_row < H - 1)
                            tile[c][actual_tr + 2][actual_tc] = input[(c * H * W) + ((image_row + 1) * W) + image_col - 1];
                        else
                            tile[c][actual_tr + 2][actual_tc] = 0.0;
                    }
                }
            }


        if(actual_tr == 0){
            for(int c=0;c<C;c++){
                if(image_row>=1)
                    tile[c][actual_tr][actual_tc+1] = input[(c*H*W)+((image_row-1)*W)+image_col] ;
                else
                    tile[c][actual_tr][actual_tc+1] = 0.0;
            }

            if(actual_tc==0){
                for(int c = 0; c < C; c++){
                    if (image_col >= 1 && image_row >= 1)
                        tile[c][actual_tr][actual_tc] = input[(c * H * W) + ((image_row - 1) * W) + image_col - 1];
                    else
                        tile[c][actual_tr][actual_tc] = 0.0;

                }
            }
        }

        if(actual_tc == bsunpadded - 1){
            for(int c=0;c<C;c++) {
                if (image_col < W - 1)
                    tile[c][actual_tr + 1][actual_tc + 2] = input[(c * H * W) + (image_row * W) + image_col + 1];
                else
                    tile[c][actual_tr + 1][actual_tc + 2] = 0.0;
            }

            if(actual_tr==0){
                for (int c = 0; c < C; c++){
                    if (image_col < W - 1 && image_row >= 1)
                        tile[c][actual_tr][actual_tc + 2] = input[(c * H * W) + ((image_row - 1) * W) + image_col + 1];
                    else
                        tile[c][actual_tr][actual_tc + 2] = 0.0;

                }
            }
        }

        __syncthreads();

            int nrow=0, ncol=0;
           double conv_val = 0.0;
            for (int c = 0; c < C; c++) {
                for (int row = 0; row < FH; row++) {
                    for (int col = 0; col < FW; col++) {
                        double tile_val = 0.0;
                        ncol = actual_tc + col;
                        nrow = actual_tr + row;
                        double fil_val = filter[(fil_num * C * FW * FH) + (c*FW*FH)+ ((FH - row - 1) * FW) + (FW - 1 - col)];
                        tile_val = tile[c][nrow][ncol];
                        conv_val = conv_val + (fil_val * tile_val);
                    }
                }
            }

        output[(fil_num*W*H)+(image_row*W)+image_col] = conv_val;
    }
}

void C3(int W, int H, int C, int FW, int FH, int K, double* d_input, double* d_filter, double* d_output, double* h_output){
    struct timespec start, end;

    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    hipdnnTensorDescriptor_t i_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&i_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(i_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W));

    hipdnnTensorDescriptor_t o_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&o_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(o_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, H, W));

    hipdnnFilterDescriptor_t f_desc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&f_desc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(f_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW));

    hipdnnConvolutionDescriptor_t conv_desc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));

    hipdnnConvolutionFwdAlgo_t conv_algo;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, i_desc, f_desc, conv_desc, o_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv_algo));

    size_t wssize = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, i_desc, f_desc, conv_desc, o_desc, conv_algo, &wssize));
    double *d_workspace;
    hipMalloc(&d_workspace, wssize);

    size_t outsize = K*W*H* sizeof(double);
    double alpha = 1.0, beta = 0.0;

    clock_gettime(CLOCK_MONOTONIC,&start);
    checkCUDNN(hipdnnConvolutionForward(cudnn, &alpha, i_desc, d_input, f_desc, d_filter, conv_desc, conv_algo, d_workspace, wssize, &beta, o_desc, d_output));
    clock_gettime(CLOCK_MONOTONIC,&end);

    hipMemcpy(h_output, d_output, outsize, hipMemcpyDeviceToHost);

    double checksum = find_checksum(h_output);
    double time = (end.tv_sec - start.tv_sec + (end.tv_nsec - start.tv_nsec)/1E9)*1000.0;
    printf("%f,%4.3lf\n", checksum, time);

    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(i_desc);
    hipdnnDestroyTensorDescriptor(o_desc);
    hipdnnDestroyFilterDescriptor(f_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);

    hipdnnDestroy(cudnn);

}

int main(){
    int W = 4096, H = 4096, C = 3, FW = 3, FH = 3, K = 10;
    struct timespec start1, end1;
    double *d_input, *d_filter, *d_output;
    double checksum=0.0, time1 = 0.0;

    int  inputsize = H*W*C* sizeof(double);
    int filtersize = FH*FW*C*K* sizeof(double);
    int outputsize = H*W*K* sizeof(double);

    hipMalloc((void **)&d_input, inputsize);
    hipMalloc((void **)&d_filter, filtersize);
    hipMalloc((void **)&d_output, outputsize);

    double* h_input = (double*) malloc(inputsize);
    double* h_filter = (double*) malloc(filtersize);
    double* h_output = (double*) malloc(outputsize);


    for(int channel=0;channel<C;channel++){
        for(int height=0; height<H; height++){
            for(int width=0; width<W; width++){
                h_input[(channel*W*H)+(height*W)+width]= channel * (width+height);
            }
        }
    }

    for(int k=0;k<K;k++){
        for(int channel=0;channel<C;channel++){
            for(int height=0; height<FH; height++){
                for(int width=0; width<FW; width++){
                    h_filter[(k*C*FW*FH)+(channel*FW*FH)+(height*FW)+width] = (channel+k)*(width+height);
                }
            }
        }
    }

    hipMemcpy(d_input, h_input, inputsize, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, filtersize, hipMemcpyHostToDevice);

    dim3 threads1(bs, bs);
    dim3 blocks1(ceil(W/bs), ceil(H/bs), K);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    C1<<<blocks1, threads1>>>(W, H, C, FW, FH, K, d_input, d_filter, d_output);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC,&end1);
    hipMemcpy(h_output, d_output, outputsize, hipMemcpyDeviceToHost);
    time1 = (end1.tv_sec - start1.tv_sec + (end1.tv_nsec - start1.tv_nsec)/1E9)*1000.0;
    checksum = find_checksum(h_output);
    printf("%f,%4.3lf\n", checksum, time1);

    hipFree(d_output);
    hipMalloc((void **)&d_output, outputsize);

    dim3 threads2(bs, bs, K);
    dim3 blocks2(ceil(W/bs), ceil(H/bs), 1);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    C2<<<blocks2, threads2>>>(W, H, C, FW, FH, K, d_input, d_filter, d_output);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC,&end1);
    hipMemcpy(h_output, d_output, outputsize, hipMemcpyDeviceToHost);
    time1 = (end1.tv_sec - start1.tv_sec + (end1.tv_nsec - start1.tv_nsec)/1E9)*1000.0;
    checksum = find_checksum(h_output);
    printf("%f,%4.3lf\n", checksum, time1);

    hipFree(d_output);
    hipMalloc((void **)&d_output, outputsize);

    C3(W, H, C, FW, FH, K, d_input, d_filter, d_output, h_output);


    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    free(h_output);
    free(h_input);
    free(h_filter);

    return 0;

}
